#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>

using namespace std;
using namespace cv;

#define MAX_GRID_SIZE 1024
#define BLOCK_SIZE 1024
#define TILE_HEIGHT 8
#define get_mean_3(img,ind) ((((int) img[ind]) + img[ind + 1] + img[ind + 2]) / 3)

__global__ void change_brightness(uint8_t *img, int width, int height, int channels, int offset)
{
    const int img_byte_size = width * height * channels;
    const int unrolling = 4;
    const int stride = gridDim.x * blockDim.x * unrolling;
    for (int base_index = blockIdx.x * blockDim.x * unrolling; base_index < img_byte_size; base_index += stride)
    {
        int ptr = base_index + threadIdx.x * unrolling, val;
        int overflow, underflow;
        
        if (ptr + 3 < img_byte_size)
        {
            val = img[ptr];
            val += offset;
            overflow = val > 255;
            underflow = val < 0;
            val &= (underflow - 1);
            val |= (-overflow);
            img[ptr] = val & 255;


            val = img[ptr + 1];
            val += offset;
            overflow = val > 255;
            underflow = val < 0;
            val &= (underflow - 1);
            val |= (-overflow);
            img[ptr + 1] = val & 255;
            
            val = img[ptr + 2];
            val += offset;
            overflow = val > 255;
            underflow = val < 0;
            val &= (underflow - 1);
            val |= (-overflow);
            img[ptr + 2] = val & 255;

            val = img[ptr + 3];
            val += offset;
            overflow = val > 255;
            underflow = val < 0;
            val &= (underflow - 1);
            val |= (-overflow);
            img[ptr + 3] = val & 255;
        }
        else
        {
            for (int i = 0; i < img_byte_size - ptr; i++)
            {
                val = img[ptr + i];
                val += offset;
                overflow = val > 255;
                underflow = val < 0;
                val &= (underflow - 1);
                val |= (-overflow);
                img[ptr + i] = val & 255;
            }
        }
    }
}

__global__ void tiled_sobel(uint8_t *out, uint8_t *img, int width, int height, int channels, uint8_t t1, uint8_t t2)
{   // Tiling is used to minimize the amount of reads to the memory hierarchy.
    const int img_byte_size = width * height * channels;
    const int row_size = width * channels;
    const int tile_stride = gridDim.x;
    const int tile_cols = BLOCK_SIZE / TILE_HEIGHT;
    const int tiles_per_row = (width + tile_cols - 1) / tile_cols;
    const int total_tiles = ((height + TILE_HEIGHT - 3) / TILE_HEIGHT) * tiles_per_row;
    __shared__ uint8_t smem[TILE_HEIGHT + 2][tile_cols + 2];

    for (int tile_index = blockIdx.x; tile_index < total_tiles; tile_index += tile_stride)
    {   // Loop unrolling wouldn't really help much here, since this is very compute intensive.
        int y_base = (tile_index / tiles_per_row) * TILE_HEIGHT + 1;
        int x_base = (tile_index % tiles_per_row) * tile_cols;
        int ptr = y_base * row_size + x_base * channels;
        int ind_x = threadIdx.x + 1, ind_y = threadIdx.y + 1;
        int actual_tile_width = min(tile_cols, width - x_base);
        int actual_tile_height = min(TILE_HEIGHT, height - 1 - y_base);
        int location;

        // Load the middle smem part with no divergence.
        int should_calc = threadIdx.x < actual_tile_width && threadIdx.y < actual_tile_height;
        smem[ind_y][ind_x] = (-should_calc) & get_mean_3(img, threadIdx.y * row_size + threadIdx.x * channels + ptr);

        // Handle the corners of the conv with minimal divergence
        bool should_handle_corners = (threadIdx.y == 0 || threadIdx.y == TILE_HEIGHT - 1)
                                    || (threadIdx.x == 0 || threadIdx.x == tile_cols - 1);
        int corner_x = (threadIdx.x == 0) * (-1) + (threadIdx.x == tile_cols - 1) * tile_cols;
        int corner_y = (threadIdx.y == 0) * (-1) + (threadIdx.y == TILE_HEIGHT - 1) * TILE_HEIGHT;
        corner_x += (corner_x == 0) * threadIdx.x;
        corner_y += (corner_y == 0) * threadIdx.y;
        if (should_handle_corners)
        {
            location = corner_y * row_size + corner_x * channels + ptr;
            bool should_zero_out = (x_base + corner_x < 0 || x_base + corner_x >= width)
                            || (location >= img_byte_size || location < 0);
            smem[corner_y + 1][corner_x + 1] = (should_zero_out ? 0 : get_mean_3(img, location));
        }
        should_handle_corners = (threadIdx.y == 0 || threadIdx.y == TILE_HEIGHT - 1)
                                    && (threadIdx.x == 0 || threadIdx.x == tile_cols - 1);
        if (should_handle_corners)
        {
            corner_x = (threadIdx.x == 0) * (-1) + (threadIdx.x == tile_cols - 1) * tile_cols;
            corner_y = (threadIdx.y == 0) * (-1) + (threadIdx.y == TILE_HEIGHT - 1) * TILE_HEIGHT;
            location = corner_y * row_size + threadIdx.x * channels + ptr;
            smem[corner_y + 1][threadIdx.x + 1] = get_mean_3(img, location);
            location = threadIdx.y * row_size + corner_x * channels + ptr;
            smem[threadIdx.y + 1][corner_x + 1] = get_mean_3(img, location);
        }
        
        // Synchronize the block so that smem can be used safely.
        __syncthreads();

        // Apply filter without bank conflicts.
        int res_x = 0, res_y = 0;
        res_x -= smem[ind_y - 1][ind_x - 1];                // Top-Left
        res_y += smem[ind_y - 1][ind_x - 1]; 

        res_y += ((int) smem[ind_y - 1][ind_x]) << 1;       // Top-Middle
        
        res_x += smem[ind_y - 1][ind_x + 1];                // Top-Right
        res_y += smem[ind_y - 1][ind_x + 1];
        
        res_x -= ((int) smem[ind_y][ind_x - 1]) << 1;       // Middle-Left

        res_x += ((int) smem[ind_y][ind_x + 1]) << 1;       // Middle-Right

        res_x -= smem[ind_y + 1][ind_x - 1];                // Bottom-Left
        res_y -= smem[ind_y + 1][ind_x - 1];

        res_y -= ((int) smem[ind_y + 1][ind_x]) << 1;       // Bottom-Middle

        res_x += smem[ind_y + 1][ind_x + 1];                // Bottom-Right
        res_y -= smem[ind_y + 1][ind_x + 1];

        // Get the absolute value of the gradient components in a branchless fashion.
        int cond_x = res_x < 0;
        int cond_y = res_y < 0;
        res_x = (res_x ^ (-cond_x)) + cond_x; // Calculate the appropriate result using twos complement.
        res_y = (res_y ^ (-cond_y)) + cond_y;

        // Clamp the result into the [0, 255] range in a branchless fashion.
        int res = res_x + res_y;
        uint8_t set_all = res > 255;
        if (should_calc)
        {
            uint8_t writeback = (-set_all) | ((uint8_t) res);
            uint8_t check_t1 = writeback <= t1;     // Threshold the output in a branchless fashion.
            uint8_t check_t2 = writeback >= t2;
            writeback &= (check_t1 - 1);
            writeback |= (-check_t2);
            out[(y_base - 1 + threadIdx.y) * width + x_base + threadIdx.x] = writeback;
        }
        __syncthreads();
    }
}

// __global__ void sobel(uint8_t *out, uint8_t *img, int width, int height, int channels)
// {
//     const int stride = gridDim.x * BLOCK_SIZE;
//     const int img_size = width * height * channels;
//     extern __shared__ uint8_t cache_data[];
    
//     const int row_byte_offset = width * channels;
//     const int stopping_bound = img_size - row_byte_offset;
//     int load_base = blockIdx.x * BLOCK_SIZE + row_byte_offset;
//     int load_cache_stride = row_byte_offset + 2 * channels;
//     while (load_base < stopping_bound)
//     {
//         if (load_base + threadIdx.x < stopping_bound)
//         {
//             int ind = threadIdx.x + channels;
//             // Load data into shared memory for conv.
//             if (threadIdx.x < 3)       // Minimally divergent behaviour.
//             {       // Corner case: doesn't even deal with bank conflicts!
//                 cache_data[threadIdx.x] = img[load_base - channels + threadIdx.x - row_byte_offset];
//                 cache_data[BLOCK_SIZE + threadIdx.x + channels] =
//                         img[load_base + threadIdx.x + BLOCK_SIZE - row_byte_offset];
//             }
//             cache_data[ind] = img[load_base + threadIdx.x - row_byte_offset];
//             if (threadIdx.x < 3)       // Minimally divergent behaviour.
//             {       // Corner case: doesn't even deal with bank conflicts!
//                 cache_data[load_cache_stride + threadIdx.x] = img[load_base - channels + threadIdx.x];
//                 cache_data[load_cache_stride + BLOCK_SIZE + threadIdx.x + channels] =
//                         img[load_base + threadIdx.x + BLOCK_SIZE];
//             }
//             cache_data[load_cache_stride + ind] = img[load_base + threadIdx.x];
//             if (threadIdx.x < 3)       // Minimally divergent behaviour.
//             {       // Corner case: doesn't even deal with bank conflicts!
//                 cache_data[load_cache_stride * 2 + threadIdx.x] =
//                         img[load_base - channels + threadIdx.x + row_byte_offset];
//                 cache_data[load_cache_stride * 2 + BLOCK_SIZE + threadIdx.x + channels] =
//                         img[load_base + threadIdx.x + BLOCK_SIZE + row_byte_offset];
//             }
//             cache_data[load_cache_stride * 2 + ind] = img[load_base + threadIdx.x + row_byte_offset];

//             // Calculate the result. Has no bank conflicts!
//             int res_x = 0;
//             int res_y = (((int) cache_data[ind]) - cache_data[ind + load_cache_stride * 2]) << 1;
//             if (((load_base + threadIdx.x) / channels) % width != 0)
//             {
//                 res_y += ((int) cache_data[ind - channels]) - cache_data[ind + load_cache_stride * 2 - channels];
//                 res_x -= ((int) cache_data[ind + load_cache_stride - channels]) << 1;
//                 res_x -= ((int) cache_data[ind - channels]) + cache_data[ind + load_cache_stride - channels];
//             }
//             if (((load_base + threadIdx.x) / channels) % width != width - 1)
//             {
//                 res_y += ((int) cache_data[ind + channels]) - cache_data[ind + load_cache_stride * 2 + channels];
//                 res_x += ((int) cache_data[ind + load_cache_stride + channels]) << 1;
//                 res_x += ((int) cache_data[ind + channels]) + cache_data[ind + load_cache_stride * 2 + channels];
//             }
//             // Get the absolute value of the gradient components in a branchless fashion.
//             int cond_x = res_x < 0;
//             int cond_y = res_y < 0;
//             res_x = (res_x ^ cond_x) + cond_x;  // Calculate the appropriate result using twos complement.
//             res_y = (res_y ^ cond_y) + cond_y;

//             // Clamp the result into the [0, 255] range in a branchless fashion.
//             int res = res_x + res_y;
//             uint8_t set_all = res > 255;
//             out[load_base + threadIdx.x - row_byte_offset] = (-set_all) | ((uint8_t) res);
//         }
//         load_base += stride;
//     }
// }

int main(int argc, char *argv[])
{
    if (argc < 7) {
        cout << "Please specify the desired origin and destination (brightness and Sobel) paths." << endl;
        return 0;
    }
    hipError_t cudaerr;
    string img_path;
    img_path = argv[1];
    // img_path = "./Test_Images/01_7680x4320.jpg";
    // cin >> img_path;
    int temp_threshold_1, temp_threshold_2, temp_brigtness_change;
    try {
        temp_threshold_1 = atoi(argv[3]);
        temp_threshold_2 = atoi(argv[4]);
        temp_brigtness_change = atoi(argv[5]);
    } catch (int e) {
        cout << "Please specify a valid number for threshold and brightness change amount." << endl;
        return 0;
    }
    const int threshold_1 = temp_threshold_1, threshold_2 = temp_threshold_2;
    const int brigtness_change = temp_brigtness_change;

    Mat input_img = imread(img_path, IMREAD_COLOR);

    uint8_t *sys_img;
    uint8_t *result;
    int img_size = input_img.rows * input_img.cols * input_img.channels();
    int padding = input_img.cols * input_img.channels();

    int padded_size = img_size + padding * (1 + input_img.channels());
    hipMalloc(&sys_img, padded_size);
    hipMemcpy(sys_img + padding, input_img.data, img_size, hipMemcpyHostToDevice);

    auto t1 = chrono::high_resolution_clock::now();
    int grid_size = min((img_size + BLOCK_SIZE - 1) / BLOCK_SIZE, MAX_GRID_SIZE);
    change_brightness<<<grid_size, BLOCK_SIZE>>>(sys_img, input_img.cols,
                                input_img.rows + 1 + input_img.channels(), input_img.channels(), brigtness_change);
    
    cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
    {
        cerr << "Brightness change failed with error \"" << hipGetErrorString(cudaerr) << "\"." << endl;
        return 1;
    }
    auto res = chrono::high_resolution_clock::now() - t1;
    
    hipMemcpy(input_img.data, sys_img, img_size, hipMemcpyDeviceToHost);

    hipMemset(sys_img, 0, padding);
    hipMemset(sys_img + padding + img_size, 0, padding * input_img.channels());
    hipMalloc(&result, input_img.rows * input_img.cols);

    t1 = chrono::high_resolution_clock::now();
    grid_size = min((input_img.rows * input_img.cols + BLOCK_SIZE - 1) / BLOCK_SIZE, MAX_GRID_SIZE);
    dim3 block_size;
    block_size.x = BLOCK_SIZE / TILE_HEIGHT;
    block_size.y = TILE_HEIGHT;
    tiled_sobel<<<grid_size, block_size>>>(result, sys_img,
                    input_img.cols, input_img.rows + 2, input_img.channels(), threshold_1, threshold_2);
    Mat output_img(input_img.rows, input_img.cols, CV_8UC1);
    cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
    {
        cerr << "Sobel launch failed with error \"" << hipGetErrorString(cudaerr) << "\"." << endl;
        return 1;
    }
    res += chrono::high_resolution_clock::now() - t1;
    hipMemcpy(output_img.data, result, input_img.rows * input_img.cols, hipMemcpyDeviceToHost);
    hipFree(sys_img);
    hipFree(result);

    long long microseconds = chrono::duration_cast<chrono::microseconds>(res).count();
    cout << "Execution Time: " << microseconds << " microseconds" << endl;

    imwrite(argv[2], input_img);
    // imwrite("./Result_Images/output_cuda_brightness.png", input_img);
    imwrite(argv[3], output_img);
    // imwrite("./Result_Images/output_cuda.png", output_img);
}
